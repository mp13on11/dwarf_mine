#include "hip/hip_runtime.h"
#include "CudaProxy.h"

#include "common/Utils.h"

#include <cuda-utils/ErrorHandling.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <cassert>
#include <cstdio>

const int THREADS_PER_BLOCK = 64;
const int THREADS_PER_BLOCK_RANDOM_KERNEL = 128;


__global__ void setupStateForRandom(hiprandState* state, size_t* seeds);
__global__ void setupStateForRandom(hiprandState* states, float* randomValues, size_t numberOfRandomValues);
__global__ void simulateGamePreRandom(size_t reiterations, size_t numberOfBlocks, float* randomValues, size_t numberOfPlayfields, const Field* playfields, Player currentPlayer, Result* results);


__global__ void testNumberOfMarkedFields(size_t* sum, const bool* playfield);
__global__ void testRandomNumber(float fakedRandom, size_t maximum, size_t* randomNumberResult);
__global__ void testDoStep(hiprandState* deviceState, Field* playfield, Player currentPlayer, float fakedRandom);
__global__ void testExpandLeaf(hiprandState* deviceState, Field* playfield, Player currentPlayer, size_t* wins, size_t* visits);

void gameSimulationPreRandom(size_t numberOfBlocks, size_t iterations, float* randomValues, size_t numberOfRandomValues, size_t numberOfPlayfields, const Field* playfields, Player currentPlayer, Result* results)
{
    hiprandState* deviceStates;
    hipMalloc(&deviceStates, sizeof(hiprandState) * THREADS_PER_BLOCK_RANDOM_KERNEL);
    setupStateForRandom<<< 1, THREADS_PER_BLOCK_RANDOM_KERNEL >>>(deviceStates, randomValues, numberOfRandomValues);
    CudaUtils::checkState();

    simulateGamePreRandom <<< numberOfBlocks, THREADS_PER_BLOCK >>> (iterations, numberOfBlocks, randomValues, numberOfPlayfields, playfields, currentPlayer, results);
    CudaUtils::checkState();
}

void gameSimulationPreRandomStreamed(size_t numberOfBlocks, size_t iterations, float* randomValues, size_t numberOfRandomValues, size_t numberOfPlayfields, const Field* playfields, Player currentPlayer, Result* results, hipStream_t stream, size_t streamSeed)
{
    hiprandState* deviceStates;
    hipMalloc(&deviceStates, sizeof(hiprandState) * THREADS_PER_BLOCK_RANDOM_KERNEL);

    setupStateForRandom<<< 1, THREADS_PER_BLOCK_RANDOM_KERNEL, 0, stream >>>(deviceStates, randomValues, numberOfRandomValues);
    CudaUtils::checkState();

    simulateGamePreRandom <<< numberOfBlocks, THREADS_PER_BLOCK, 0, stream >>> (iterations, numberOfBlocks, randomValues, numberOfPlayfields, playfields, currentPlayer, results);
    CudaUtils::checkState();
}

void setupSeedForTest(size_t numberOfBlocks, hiprandState* deviceStates)
{
    size_t* seed;
    
    hipMalloc(&seed, sizeof(size_t) * numberOfBlocks);
    hipMalloc(&deviceStates, sizeof(hiprandState) * numberOfBlocks);
    
    setupStateForRandom <<< numberOfBlocks, 1 >>>(deviceStates, seed);
    
    CudaUtils::checkState();
}

void testDoStepProxy(Field* playfield, Player currentPlayer, float fakedRandom)
{
    hiprandState* deviceStates = NULL;
    size_t numberOfBlocks = 1;
    setupSeedForTest(numberOfBlocks, deviceStates);

    testDoStep <<< numberOfBlocks, THREADS_PER_BLOCK >>>(deviceStates, playfield, currentPlayer, fakedRandom);
    CudaUtils::checkState();    
}

void testNumberOfMarkedFieldsProxy(size_t* sum, const bool* playfield)
{
    testNumberOfMarkedFields<<< 1, THREADS_PER_BLOCK >>>(sum, playfield);
    CudaUtils::checkState();
}

void testRandomNumberProxy(float fakedRandom, size_t maximum, size_t* randomMoveIndex)
{
    testRandomNumber<<< 1, 1 >>> (fakedRandom, maximum, randomMoveIndex);
    CudaUtils::checkState();
}

void testExpandLeafProxy(size_t dimension, Field* playfield, Player currentPlayer, size_t* wins, size_t* visits)
{
    hiprandState* deviceStates = NULL;
    size_t numberOfBlocks = 1;
    setupSeedForTest(numberOfBlocks, deviceStates);
    
    testExpandLeaf <<< numberOfBlocks, THREADS_PER_BLOCK >>>(deviceStates, playfield, currentPlayer, wins, visits);
    CudaUtils::checkState();
}
