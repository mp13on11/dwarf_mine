#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "MatrixMultiplication.h"

const size_t BLOCK_SIZE = DEFAULT_BLOCK_SIZE;

__device__ int div_ceil_d(int x, int y)
{
//    return 1 + ((x - 1) / y);
//    return (x + y - 1) / y;

    return (x % y) ? x / y + 1 : x / y;
}

struct Matrix
{
    int cols;
    int rows;
    int stride;
    float* data;
};

__device__ void setElement(Matrix m, int row, int col, float value)
{
    if (row >= m.rows || col >= m.cols) return;
    m.data[(m.stride * row) + col] = value;
}

__device__ float getElement(Matrix m, int row, int col)
{
    if (row >= m.rows || col >= m.cols) return 0;
    return m.data[(m.stride * row) + col];
}

__device__ Matrix getSubMatrix(Matrix m, int blockRow, int blockColumn)
{
    Matrix n;
    n.rows = ((blockRow+1)*blockDim.x > m.rows) ? (m.rows - blockRow*blockDim.x) : blockDim.x;
    n.cols = ((blockColumn+1)*blockDim.x > m.cols) ? (m.cols - blockColumn*blockDim.x) : blockDim.x;
    n.stride = m.stride;
    n.data = &m.data[blockRow * m.stride * blockDim.x + blockColumn * blockDim.x];
    return n;
}


__global__ void gemmKernel(int m, int n, int k, float* left, float* right, float* out)
{
    Matrix leftMatrix;
    leftMatrix.rows = m;
    leftMatrix.cols = k;
    leftMatrix.stride = k;
    leftMatrix.data = left;

    Matrix rightMatrix;
    rightMatrix.rows = k;
    rightMatrix.cols = n;
    rightMatrix.stride = n;
    rightMatrix.data = right;

    Matrix outMatrix;
    outMatrix.rows = m;
    outMatrix.cols = n;
    outMatrix.stride = n;
    outMatrix.data = out;

    int blockRow = blockIdx.y;
    int blockColumn = blockIdx.x;

    int row = threadIdx.y;
    int col = threadIdx.x;

    Matrix outSub = getSubMatrix(outMatrix, blockRow, blockColumn);

    float sum = 0.0f;

    for (int block = 0, end= div_ceil_d(leftMatrix.cols, blockDim.x); block < end ; ++block)
    {
        __shared__ float leftSub_s[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float rightSub_s[BLOCK_SIZE][BLOCK_SIZE];

        Matrix leftSub = getSubMatrix(leftMatrix, blockRow, block);
        Matrix rightSub = getSubMatrix(rightMatrix, block, blockColumn);

        leftSub_s[row][col] = getElement(leftSub, row, col);
        rightSub_s[row][col] = getElement(rightSub, row, col);

        __syncthreads();

        for (int i = 0; i < blockDim.x; ++i)
        {
            sum += leftSub_s[row][i] * rightSub_s[i][col];
        }
    }

    setElement(outSub, row, col, sum);
}
