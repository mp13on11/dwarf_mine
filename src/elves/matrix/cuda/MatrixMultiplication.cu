#include "hip/hip_runtime.h"
#include "MatrixMultiplication.h"
#include "common/Utils.h"

#include <cuda-utils/ErrorHandling.h>
#include <hip/hip_runtime.h>
#include <iostream>

//kernel declaration
__global__ void gemmKernel(int m, int n, int k, float* left, float* right, float* out);

//kernel calling function
void gemm(int m, int n, int k, float* left, float* right, float* out, int blockSize)
{
	using namespace std;

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(gemmKernel), hipFuncCachePreferL1);

    dim3 dimGrid(div_ceil(n, blockSize), div_ceil(m, blockSize));
    dim3 dimBlock(blockSize, blockSize);
    gemmKernel <<< dimGrid, dimBlock >>>(m, n, k, left, right, out);
    CudaUtils::checkState();
}
