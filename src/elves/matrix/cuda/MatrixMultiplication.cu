#include "hip/hip_runtime.h"
/*****************************************************************************
* Dwarf Mine - The 13-11 Benchmark
*
* Copyright (c) 2013 Bünger, Thomas; Kieschnick, Christian; Kusber,
* Michael; Lohse, Henning; Wuttke, Nikolai; Xylander, Oliver; Yao, Gary;
* Zimmermann, Florian
*
* Permission is hereby granted, free of charge, to any person obtaining
* a copy of this software and associated documentation files (the
* "Software"), to deal in the Software without restriction, including
* without limitation the rights to use, copy, modify, merge, publish,
* distribute, sublicense, and/or sell copies of the Software, and to
* permit persons to whom the Software is furnished to do so, subject to
* the following conditions:
*
* The above copyright notice and this permission notice shall be
* included in all copies or substantial portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
* EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
* MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
* IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
* CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
* TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
* SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*****************************************************************************/

#include "MatrixMultiplication.h"
#include "common/Utils.h"

#include <cuda-utils/ErrorHandling.h>
#include <hip/hip_runtime.h>
#include <iostream>

//kernel declaration
__global__ void gemmKernel(int m, int n, int k, float* left, float* right, float* out);

//kernel calling function
void gemm(int m, int n, int k, float* left, float* right, float* out, int blockSize)
{
	using namespace std;

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(gemmKernel), hipFuncCachePreferL1);

    dim3 dimGrid(div_ceil(n, blockSize), div_ceil(m, blockSize));
    dim3 dimBlock(blockSize, blockSize);
    gemmKernel <<< dimGrid, dimBlock >>>(m, n, k, left, right, out);
    CudaUtils::checkState();
