#include "kernel.cuh"
#include "Number.cuh"
#include "Factorize.h"
#include "common/Utils.h"

const size_t BLOCK_SIZE = 64;

void megaWrapper(uint32_t* logs_d, const uint32_t* factorBase_d, const size_t factorBaseSize, const uint32_t* start, const uint32_t* end, const uint32_t intervalLength)
{
    size_t numThreads = div_ceil(intervalLength, (uint32_t)NUMBERS_PER_THREAD);
    size_t numBlocks = div_ceil(numThreads, BLOCK_SIZE);
	Number* start_d; 
	hipMalloc(&start_d, sizeof(uint32_t)*NUM_FIELDS);
	hipMemcpy(start_d, start, 10, hipMemcpyHostToDevice);
	Number* end_d; 
	hipMalloc(&end_d, sizeof(uint32_t)*NUM_FIELDS);
	hipMemcpy(end_d, end, 10, hipMemcpyHostToDevice);
	
	megaKernel<<<numBlocks, BLOCK_SIZE>>>(logs_d, factorBase_d, (int)factorBaseSize, start_d, end_d, intervalLength);
	hipFree(start_d);
	hipFree(end_d);
}