#include "hip/hip_runtime.h"
#include "MonteCarloTreeSearch.h"

#include "common/Utils.h"

#include <cuda-utils/ErrorHandling.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <cassert>
#include <cstdio>

const int THREADS_PER_BLOCK = 64;

__global__ void setupStateForRandom(hiprandState* state, size_t* seeds);
__global__ void simulateGameLeaf(hiprandState* deviceState, Field* playfield, Player currentPlayer, size_t* wins, size_t* visits);
__global__ void simulateGame(size_t reiterations, hiprandState* deviceStates, size_t numberOfPlayfields, Field* playfields, Player currentPlayer, OthelloResult* results);

__global__ void testDoStep(hiprandState* deviceState, Field* playfield, Player currentPlayer, float fakedRandom);
__global__ void testSimulateGameLeaf(hiprandState* deviceState, Field* playfield, Player currentPlayer, size_t* wins, size_t* visits);

void gameSimulation(size_t numberOfBlocks, size_t iterations, size_t* seeds, size_t numberOfPlayfields, Field* playfields, Player currentPlayer, OthelloResult* results)
{
    hiprandState* deviceStates;
    hipMalloc(&deviceStates, sizeof(hiprandState) * numberOfBlocks);
    
    setupStateForRandom <<< numberOfBlocks, 1 >>> (deviceStates, seeds);
    CudaUtils::checkState();
    
    simulateGame <<< numberOfBlocks, THREADS_PER_BLOCK >>> (size_t(ceil(iterations * 1.0 / numberOfBlocks)), deviceStates, numberOfPlayfields, playfields, currentPlayer, results);
    CudaUtils::checkState();
}

void leafSimulation(size_t reiterations, size_t dimension, Field* playfield, Player currentPlayer, size_t* moveX, size_t* moveY, size_t* wins, size_t* visits)
{
    hiprandState* deviceStates;
    hipMalloc(&deviceStates, sizeof(hiprandState) * 1 * THREADS_PER_BLOCK);
    
    setupStateForRandom <<< 1, THREADS_PER_BLOCK >>>(deviceStates, 0ULL);
    CudaUtils::checkState();

    simulateGameLeaf <<< 1, THREADS_PER_BLOCK >>>(deviceStates, playfield, currentPlayer, wins, visits);
    CudaUtils::checkState();
}

void testBySimulateSingeStep(Field* playfield, Player currentPlayer, float fakedRandom)
{
    hiprandState* deviceStates;
    hipMalloc(&deviceStates, sizeof(hiprandState) * 1 * THREADS_PER_BLOCK);
    setupStateForRandom <<< 1, THREADS_PER_BLOCK >>>(deviceStates, 0ULL);
    CudaUtils::checkState();

    testDoStep <<< 1, THREADS_PER_BLOCK >>>(deviceStates, playfield, currentPlayer, fakedRandom);
    CudaUtils::checkState();    
}

void testByLeafSimulation(size_t dimension, Field* playfield, Player currentPlayer, size_t* wins, size_t* visits)
{
    hiprandState* deviceStates;
    hipMalloc(&deviceStates, sizeof(hiprandState) * 1 * THREADS_PER_BLOCK);
    setupStateForRandom <<< 1, THREADS_PER_BLOCK >>>(deviceStates, 0UL);
    CudaUtils::checkState();
    
    testSimulateGameLeaf <<< 1, THREADS_PER_BLOCK >>>(deviceStates, playfield, currentPlayer, wins, visits);
    CudaUtils::checkState();
}