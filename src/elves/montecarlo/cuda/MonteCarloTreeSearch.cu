#include "hip/hip_runtime.h"
#include "MonteCarloTreeSearch.h"

#include "common/Utils.h"

#include <cuda-utils/ErrorHandling.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

const int NUMBER_OF_BLOCKS = 1;
const int THREADS_PER_BLOCK = 64;

__global__ void setupStateForRandom(hiprandState* state, unsigned long seed);
__global__ void simulateGameLeaf(hiprandState* deviceState, Field* playfield, Player currentPlayer, size_t* wins, size_t* visits);
__global__ void simulateGame(hiprandState* deviceStates, size_t fieldDimension, Field* playfields, Player currentPlayer, size_t* wins, size_t* visits);

__global__ void testDoStep(hiprandState* deviceState, Field* playfield, Player currentPlayer, float fakedRandom);
__global__ void testSimulateGameLeaf(hiprandState* deviceState, Field* playfield, Player currentPlayer, size_t* wins, size_t* visits);

void leafSimulation(size_t reiterations, size_t dimension, Field* playfield, Player currentPlayer, size_t* moveX, size_t* moveY, size_t* wins, size_t* visits)
{
    hiprandState* deviceStates;
    hipMalloc(&deviceStates, sizeof(hiprandState));
    setupStateForRandom <<< NUMBER_OF_BLOCKS, THREADS_PER_BLOCK >>>(deviceStates, 0ULL);
    simulateGameLeaf <<< NUMBER_OF_BLOCKS, THREADS_PER_BLOCK >>>(deviceStates, playfield, currentPlayer, wins, visits);
    CudaUtils::checkState();
}

void testBySimulateSingeStep(Field* playfield, Player currentPlayer, float fakedRandom)
{
    hiprandState* deviceStates;
    hipMalloc(&deviceStates, sizeof(hiprandState));
    setupStateForRandom <<< NUMBER_OF_BLOCKS, THREADS_PER_BLOCK >>>(deviceStates, 0ULL);
    testDoStep <<< NUMBER_OF_BLOCKS, THREADS_PER_BLOCK >>>(deviceStates, playfield, currentPlayer, fakedRandom);
    CudaUtils::checkState();    
}

void testByLeafSimulation(size_t dimension, Field* playfield, Player currentPlayer, size_t* wins, size_t* visits)
{
    hiprandState* deviceStates;
    hipMalloc(&deviceStates, sizeof(hiprandState));
    setupStateForRandom <<< NUMBER_OF_BLOCKS, THREADS_PER_BLOCK >>>(deviceStates, 0ULL);
    testSimulateGameLeaf <<< NUMBER_OF_BLOCKS, THREADS_PER_BLOCK >>>(deviceStates, playfield, currentPlayer, wins, visits);
    CudaUtils::checkState();
}