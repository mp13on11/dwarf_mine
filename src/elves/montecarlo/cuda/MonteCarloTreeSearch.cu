#include "hip/hip_runtime.h"
#include "MonteCarloTreeSearch.h"

#include "common/Utils.h"

#include <cuda-utils/ErrorHandling.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <cassert>
#include <cstdio>

const int NUMBER_OF_BLOCKS = 1;
const int THREADS_PER_BLOCK = 64;

__global__ void setupStateForRandom(hiprandState* state, unsigned long seed);
__global__ void simulateGameLeaf(hiprandState* deviceState, Field* playfield, Player currentPlayer, size_t* wins, size_t* visits);
__global__ void simulateGame(size_t reiterations, hiprandState* deviceStates, size_t numberOfPlayfields, Field* playfields, Player currentPlayer, OthelloResult* results);

__global__ void testDoStep(hiprandState* deviceState, Field* playfield, Player currentPlayer, float fakedRandom);
__global__ void testSimulateGameLeaf(hiprandState* deviceState, Field* playfield, Player currentPlayer, size_t* wins, size_t* visits);

static size_t seed = 70;

void gameSimulation(size_t reiterations, size_t numberOfPlayfields, Field* playfields, Player currentPlayer, OthelloResult* results)
{
    hiprandState* deviceStates;
    hipMalloc(&deviceStates, sizeof(hiprandState) * NUMBER_OF_BLOCKS);
    //size_t seed = time(NULL);
    std::cout<<"Seed: "<< seed << std::endl;
    setupStateForRandom <<< NUMBER_OF_BLOCKS, 1 >>> (deviceStates, seed);
    seed++;
    simulateGame <<< NUMBER_OF_BLOCKS, THREADS_PER_BLOCK >>> (reiterations, deviceStates, numberOfPlayfields, playfields, currentPlayer, results);
    CudaUtils::checkState();
}

void leafSimulation(size_t reiterations, size_t dimension, Field* playfield, Player currentPlayer, size_t* moveX, size_t* moveY, size_t* wins, size_t* visits)
{
    hiprandState* deviceStates;
    hipMalloc(&deviceStates, sizeof(hiprandState));
    setupStateForRandom <<< 1, THREADS_PER_BLOCK >>>(deviceStates, 0ULL);
    simulateGameLeaf <<< 1, THREADS_PER_BLOCK >>>(deviceStates, playfield, currentPlayer, wins, visits);
    CudaUtils::checkState();
}

void testBySimulateSingeStep(Field* playfield, Player currentPlayer, float fakedRandom)
{
    hiprandState* deviceStates;
    hipMalloc(&deviceStates, sizeof(hiprandState));
    setupStateForRandom <<< 1, THREADS_PER_BLOCK >>>(deviceStates, 0ULL);
    testDoStep <<< 1, THREADS_PER_BLOCK >>>(deviceStates, playfield, currentPlayer, fakedRandom);
    CudaUtils::checkState();    
}

void testByLeafSimulation(size_t dimension, Field* playfield, Player currentPlayer, size_t* wins, size_t* visits)
{
    hiprandState* deviceStates;
    hipMalloc(&deviceStates, sizeof(hiprandState));
    setupStateForRandom <<< 1, THREADS_PER_BLOCK >>>(deviceStates, 0UL);
    testSimulateGameLeaf <<< 1, THREADS_PER_BLOCK >>>(deviceStates, playfield, currentPlayer, wins, visits);
    CudaUtils::checkState();
}