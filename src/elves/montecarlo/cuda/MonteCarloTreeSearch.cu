#include "hip/hip_runtime.h"
#include "MonteCarloTreeSearch.h"

#include "common/Utils.h"

#include <cuda-utils/ErrorHandling.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <cassert>
#include <cstdio>

const int THREADS_PER_BLOCK = 64;

__global__ void setupStateForRandom(hiprandState* state, size_t* seeds);
__global__ void setupStateForRandom(hiprandState* states, float* randomValues, size_t numberOfRandomValues, size_t streamSeed = 0);
__global__ void simulateGame(size_t reiterations, hiprandState* deviceStates, size_t numberOfPlayfields, const Field* playfields, Player currentPlayer, OthelloResult* results);
__global__ void simulateGamePreRandom(size_t reiterations, float* randomValues, size_t numberOfRandomValues, size_t numberOfPlayfields, const Field* playfields, Player currentPlayer, OthelloResult* results);
__global__ void simulateGamePreRandom(size_t reiterations, size_t numberOfBlocks, float* randomValues, size_t numberOfPlayfields, const Field* playfields, Player currentPlayer, OthelloResult* results);

__global__ void testNumberOfMarkedFields(size_t* sum, const bool* playfield);
__global__ void testRandomNumber(float fakedRandom, size_t maximum, size_t* randomNumberResult);
__global__ void testDoStep(hiprandState* deviceState, Field* playfield, Player currentPlayer, float fakedRandom);
__global__ void testExpandLeaf(hiprandState* deviceState, Field* playfield, Player currentPlayer, size_t* wins, size_t* visits);

void gameSimulation(size_t numberOfBlocks, size_t iterations, size_t* seeds, size_t numberOfPlayfields, const Field* playfields, Player currentPlayer, OthelloResult* results)
{
    hiprandState* deviceStates = NULL;
    hipMalloc(&deviceStates, sizeof(hiprandState) * numberOfBlocks);
    
    setupStateForRandom <<< numberOfBlocks, 1 >>> (deviceStates, seeds);
    CudaUtils::checkState();
    
    simulateGame <<< numberOfBlocks, THREADS_PER_BLOCK >>> (size_t(ceil(iterations * 1.0 / numberOfBlocks)), deviceStates, numberOfPlayfields, playfields, currentPlayer, results);
    CudaUtils::checkState();
}

void gameSimulationPreRandom(size_t numberOfBlocks, size_t iterations, float* randomValues, size_t numberOfPlayfields, const Field* playfields, Player currentPlayer, OthelloResult* results)
{
    simulateGamePreRandom <<< numberOfBlocks, THREADS_PER_BLOCK >>> (size_t(ceil(iterations * 1.0 / numberOfBlocks)), numberOfBlocks, randomValues, numberOfPlayfields, playfields, currentPlayer, results);
    CudaUtils::checkState();
}

void gameSimulationPreRandom(size_t numberOfBlocks, size_t iterations, float* randomValues, size_t numberOfRandomValues, size_t numberOfPlayfields, const Field* playfields, Player currentPlayer, OthelloResult* results)
{
    hiprandState* deviceStates = NULL;
    hipMalloc(&deviceStates, sizeof(hiprandState) * 128);
    setupStateForRandom<<< 1, 128 >>>(deviceStates, randomValues, numberOfRandomValues);
    CudaUtils::checkState();

    simulateGamePreRandom <<< numberOfBlocks, THREADS_PER_BLOCK >>> (iterations, numberOfBlocks, randomValues, numberOfPlayfields, playfields, currentPlayer, results);
    CudaUtils::checkState();
}

void gameSimulationPreRandom(size_t numberOfBlocks, size_t iterations, float* randomValues, size_t numberOfRandomValues, size_t numberOfPlayfields, const Field* playfields, Player currentPlayer, OthelloResult* results, hipStream_t stream, size_t streamSeed)
{
    hiprandState* deviceStates = NULL;
    hipMalloc(&deviceStates, sizeof(hiprandState) * 128);
    setupStateForRandom<<< 1, 128, 0, stream >>>(deviceStates, randomValues, numberOfRandomValues, streamSeed);
    CudaUtils::checkState();

    simulateGamePreRandom <<< numberOfBlocks, THREADS_PER_BLOCK, 0, stream >>> (iterations, numberOfBlocks, randomValues, numberOfPlayfields, playfields, currentPlayer, results);
    CudaUtils::checkState();
}

void gameSimulationStreamed(size_t numberOfBlocks, size_t iterations, size_t* seeds, size_t numberOfPlayfields, const Field* playfields, Player currentPlayer, OthelloResult* results, hipStream_t stream)
{
    hiprandState* deviceStates = NULL;
    hipMalloc(&deviceStates, sizeof(hiprandState) * numberOfBlocks);
    
    setupStateForRandom <<< numberOfBlocks, 1, 0, stream >>> (deviceStates, seeds);
    //CudaUtils::checkState();
    
    simulateGame <<< numberOfBlocks, THREADS_PER_BLOCK, 0, stream >>> (size_t(ceil(iterations * 1.0 / numberOfBlocks)), deviceStates, numberOfPlayfields, playfields, currentPlayer, results);
    //CudaUtils::checkState();
}

void setupSeedForTest(size_t numberOfBlocks, hiprandState* deviceStates)
{
    size_t* seed;
    
    hipMalloc(&seed, sizeof(size_t) * numberOfBlocks);
    hipMalloc(&deviceStates, sizeof(hiprandState) * numberOfBlocks);
    
    setupStateForRandom <<< numberOfBlocks, 1 >>>(deviceStates, seed);
    
    CudaUtils::checkState();
}

void testDoStepProxy(Field* playfield, Player currentPlayer, float fakedRandom)
{
    hiprandState* deviceStates = NULL;
    size_t numberOfBlocks = 1;
    setupSeedForTest(numberOfBlocks, deviceStates);

    testDoStep <<< numberOfBlocks, THREADS_PER_BLOCK >>>(deviceStates, playfield, currentPlayer, fakedRandom);
    CudaUtils::checkState();    
}

void testNumberOfMarkedFieldsProxy(size_t* sum, const bool* playfield)
{
    testNumberOfMarkedFields<<< 1, THREADS_PER_BLOCK >>>(sum, playfield);
    CudaUtils::checkState();
}

void testRandomNumberProxy(float fakedRandom, size_t maximum, size_t* randomMoveIndex)
{
    testRandomNumber<<< 1, 1 >>> (fakedRandom, maximum, randomMoveIndex);
    CudaUtils::checkState();
}

void testExpandLeafProxy(size_t dimension, Field* playfield, Player currentPlayer, size_t* wins, size_t* visits)
{
    hiprandState* deviceStates = NULL;
    size_t numberOfBlocks = 1;
    setupSeedForTest(numberOfBlocks, deviceStates);
    
    testExpandLeaf <<< numberOfBlocks, THREADS_PER_BLOCK >>>(deviceStates, playfield, currentPlayer, wins, visits);
    CudaUtils::checkState();
}
