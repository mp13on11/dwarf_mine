#include "hip/hip_runtime.h"
#include "MonteCarloTreeSearch.h"

#include "common/Utils.h"

#include <cuda-utils/ErrorHandling.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

const size_t BLOCK_SIZE = 1;

__global__ void setupKernel(hiprandState* state, unsigned long seed);
__global__ void computeKernel(hiprandState* deviceStates, size_t reiterations, size_t dimension, Field* playfield, size_t* moveX, size_t* moveY, size_t* wins, size_t* visits);

void compute(size_t reiterations, size_t dimension, Field* playfield, size_t* moveX, size_t* moveY, size_t* wins, size_t* visits)
{
    dim3 dimGrid(div_ceil(dimension, BLOCK_SIZE), div_ceil(dimension, BLOCK_SIZE));
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    hiprandState* deviceStates;
    hipMalloc(&deviceStates, sizeof(hiprandState));
    setupKernel <<< dimGrid, dimBlock >>>(deviceStates, 0ULL);
    computeKernel <<< dimGrid, dimBlock >>>(deviceStates, reiterations, dimension, playfield, moveX, moveY, wins, visits);
    CudaUtils::checkState();
}