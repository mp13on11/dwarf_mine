#include "hip/hip_runtime.h"
#include "MonteCarloTreeSearch.h"
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "OthelloField.h"
#include "CudaGameState.cuh"
#include "CudaMove.cuh"
#include "CudaSimulator.cuh"
#include "CudaUtil.cuh"
#include <assert.h>

__global__ void setupStateForRandom(hiprandState* state, size_t* seeds)
{
    printf("Block %d: Seed: %lu\n", blockIdx.x, seeds[blockIdx.x]);
	hiprand_init(seeds[blockIdx.x], 0, 0, &state[blockIdx.x]);
    //hiprand_init(0, 0, 0, &state[threadIdx.x]);
}


__device__ bool doStep(CudaGameState& state, CudaSimulator& simulator, size_t limit, float fakedRandom = -1)
{
    cassert(state.size == FIELD_DIMENSION * FIELD_DIMENSION, "Block %d, Thread %d detected invalid field size of %li\n", blockIdx.x, threadIdx.x, state.size);
    
    __syncthreads();
    
    simulator.calculatePossibleMoves();
    
    __syncthreads();
    
    size_t moveCount = simulator.countPossibleMoves();
    
    if (moveCount > 0)
    {
        size_t index = simulator.getRandomMoveIndex(moveCount, fakedRandom);
        cassert(index < state.size, "Block %d, Thread %d: Round %d detected unexpected move index %d for maximal playfield size %lu\n", blockIdx.x, limit, index, state.size);

        __syncthreads();

        simulator.flipEnemyCounter(index, limit);

        __syncthreads();
        
        cassert(!unchangedState(state, limit), "Block %d: %lu detected unchanged state\n", blockIdx.x, limit);
    }

    state.currentPlayer = state.getEnemyPlayer();
    return moveCount > 0;
}

const int MAXIMAL_MOVE_COUNT = 128; // an impossible condition - it would mean that for every field both players had to pass

__device__ void simulateGameLeaf(hiprandState* deviceState, CudaSimulator& simulator, CudaGameState& state, size_t* wins, size_t* visits)
{
    Player startingPlayer = state.currentPlayer;
    size_t passCounter = 0;
    size_t rounds = 0;

    __syncthreads();
    
    while (passCounter < 2)
    {
        bool passedMove = !doStep(state, simulator, rounds);
        passCounter = (passedMove ? passCounter + 1 : 0);

        cassert (rounds < MAXIMAL_MOVE_COUNT, "Detected rounds overflowing maximal count %d in %d\n", MAXIMAL_MOVE_COUNT, threadIdx.x); 
        rounds++;
    }
    __syncthreads();

    if (threadIdx.x == 0)
    {
        ++(*visits);
        if (state.isWinner(startingPlayer))
        {
            ++(*wins);
        }
    }
}

__global__ void simulateGameLeaf(hiprandState* deviceState, Field* playfield, Player currentPlayer, size_t* wins, size_t* visits)
{
    int playfieldIndex = threadIdx.x;

    __shared__ Field sharedPlayfield[FIELD_DIMENSION * FIELD_DIMENSION];
    __shared__ Field oldPlayfield[FIELD_DIMENSION * FIELD_DIMENSION];
    __shared__ bool possibleMoves[FIELD_DIMENSION*FIELD_DIMENSION];
    sharedPlayfield[playfieldIndex] = playfield[playfieldIndex];

    CudaGameState state =  { 
        sharedPlayfield,
        oldPlayfield, 
        possibleMoves, 
        FIELD_DIMENSION * FIELD_DIMENSION, 
        FIELD_DIMENSION, 
        currentPlayer 
    };
    CudaSimulator simulator(&state, deviceState);
    simulateGameLeaf(deviceState, simulator, state, wins, visits);
}

__global__ void simulateGame(size_t reiterations, hiprandState* deviceStates, size_t numberOfPlayfields, Field* playfields, Player currentPlayer, OthelloResult* results)
{
    int playfieldIndex = threadIdx.x;

    for (size_t i = 0; i < reiterations; ++i)
    {
        size_t node = randomNumber(deviceStates, numberOfPlayfields);

        __shared__ Field sharedPlayfield[FIELD_DIMENSION * FIELD_DIMENSION];
        __shared__ Field oldPlayfield[FIELD_DIMENSION * FIELD_DIMENSION];
        __shared__ bool possibleMoves[FIELD_DIMENSION*FIELD_DIMENSION];
        
        size_t playfieldOffset = FIELD_DIMENSION * FIELD_DIMENSION * node;
        sharedPlayfield[playfieldIndex] = playfields[playfieldOffset + playfieldIndex];

        CudaGameState state =  { 
            sharedPlayfield, 
            oldPlayfield,
            possibleMoves, 
            FIELD_DIMENSION * FIELD_DIMENSION, 
            FIELD_DIMENSION, 
            currentPlayer 
        };
        CudaSimulator simulator(&state, deviceStates);

        size_t wins = 0;
        size_t visits = 0;

        __syncthreads();

        simulateGameLeaf(deviceStates, simulator, state, &wins, &visits);
        
        __syncthreads();
        if (threadIdx.x == 0)
        {
            results[node].wins += wins;
            results[node].visits += visits;
        }
    }
}

__global__ void testDoStep(hiprandState* deviceState, Field* playfield, Player currentPlayer, float fakedRandom)
{
    int playfieldIndex = threadIdx.x;
    __shared__ Field sharedPlayfield[FIELD_DIMENSION * FIELD_DIMENSION];
    __shared__ Field oldPlayfield[FIELD_DIMENSION * FIELD_DIMENSION];
    __shared__ bool possibleMoves[FIELD_DIMENSION*FIELD_DIMENSION];
    sharedPlayfield[playfieldIndex] = playfield[playfieldIndex];

    CudaGameState state =  { 
        sharedPlayfield, 
        oldPlayfield,
        possibleMoves, 
        FIELD_DIMENSION * FIELD_DIMENSION, 
        FIELD_DIMENSION, 
        currentPlayer 
    };
    CudaSimulator simulator(&state, deviceState);

    doStep(state, simulator, 0, fakedRandom);

    playfield[playfieldIndex] = sharedPlayfield[playfieldIndex];
}

__global__ void testSimulateGameLeaf(hiprandState* deviceState, Field* playfield, Player currentPlayer, size_t* wins, size_t* visits)
{
    int playfieldIndex = threadIdx.x;

    __shared__ Field sharedPlayfield[FIELD_DIMENSION * FIELD_DIMENSION];
    __shared__ Field oldPlayfield[FIELD_DIMENSION * FIELD_DIMENSION];
    __shared__ bool possibleMoves[FIELD_DIMENSION*FIELD_DIMENSION];
    sharedPlayfield[playfieldIndex] = playfield[playfieldIndex];

    CudaGameState state =  { 
        sharedPlayfield, 
        oldPlayfield,
        possibleMoves, 
        FIELD_DIMENSION * FIELD_DIMENSION, 
        FIELD_DIMENSION, 
        currentPlayer 
    };
    CudaSimulator simulator(&state, deviceState);
    simulateGameLeaf(deviceState, simulator, state, wins, visits);

	playfield[playfieldIndex] = sharedPlayfield[playfieldIndex];
}