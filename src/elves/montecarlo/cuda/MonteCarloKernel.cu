#include "hip/hip_runtime.h"
#include "MonteCarloTreeSearch.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "OthelloField.h"
#include <stdio.h>

const int FIELD_DIMENSION = 8;

__global__ void setupStateForRandom(hiprandState* state, unsigned long seed)
{
	int id = 0; // threadIdx.x;
	hiprand_init(seed, id, 0, &state[id]);
}

__device__ size_t randomNumber(hiprandState* deviceStates, size_t maximum)
{
	hiprandState deviceState = deviceStates[0];
	size_t value = hiprand_uniform(&deviceState) * maximum;
	deviceStates[0] = deviceState;
    return value;
}   

typedef struct _CudaMove
{
    int x;
    int y;
} CudaMove;

typedef struct _CudaGameState
{
    Field* field;
    bool* possible;
    size_t size;
    size_t sideLength;
    Player currentPlayer;

    __device__ bool inBounds(int x, int y)
    {
        return (x >= 0 && x < sideLength && y >= 0 && y < sideLength);
    }

    __device__ bool inBounds(int i)
    {
        return (i >= 0 && i < size);
    }

    __device__ Player getEnemyPlayer()
    {
        return (currentPlayer == Black ? White : Black);
    }
} CudaGameState;


class CudaSimulator
{
private:
    size_t _playfieldIndex;
    size_t _playfieldX;
    size_t _playfieldY;
    CudaGameState* _state;
    hiprandState* _deviceState;

    __device__ bool isMaster()
    {
        return _playfieldIndex == 0;
    }
public:
    __device__ CudaSimulator(CudaGameState* state, hiprandState* deviceState)
        : _playfieldIndex(threadIdx.x), _playfieldX(_playfieldIndex % FIELD_DIMENSION), _playfieldY(_playfieldIndex / FIELD_DIMENSION),
            _state(state), _deviceState(deviceState)
        {
        }

    __device__ void calculatePossibleMoves()
    {
        _state->possible[_playfieldIndex] = false;
    
        __syncthreads();

        if (_state->field[_playfieldIndex] == Free)
        {
            findPossibleMoves( 1,  1);
            findPossibleMoves( 1,  0);
            findPossibleMoves( 1, -1);
            findPossibleMoves( 0,  1);
            
            findPossibleMoves( 0, -1);
            findPossibleMoves(-1,  1);
            findPossibleMoves(-1,  0);
            findPossibleMoves(-1, -1);
        }

        __syncthreads();
    }

    __device__ void findPossibleMoves(int directionX, int directionY)
    {
        bool look = true;
        bool foundEnemy = false;
        Player enemyPlayer = _state->getEnemyPlayer();
        int neighbourX = _playfieldX + directionX;
        int neighbourY = _playfieldY + directionY;
        while (look)
        {
            int neighbourIndex = neighbourY * FIELD_DIMENSION + neighbourX;
            if (_state->inBounds(neighbourX, neighbourY))
            {
                if (_state->field[neighbourIndex] == Free)
                {
                    _state->possible[_playfieldIndex] |= false;
                    look = false;
                }
                else if(_state->field[neighbourIndex] == enemyPlayer)
                {
                    foundEnemy = true;
                }
                else if (_state->field[neighbourIndex] == _state->currentPlayer)
                {
                    _state->possible[_playfieldIndex] |= foundEnemy;
                    look = false;
                }
            }
            else
            {
                _state->possible[_playfieldIndex] |= false;
                look = false;
            }
            neighbourX += directionX;
            neighbourY += directionY;
        }
    }

    __device__ size_t countPossibleMoves()
    {
        // __syncthreads();
     //    __shared__ size_t moves[FIELD_DIMENSION * FIELD_DIMENSION];
     //    moves[playfieldIndex] = possibleMoves[playfieldIndex] ? 1 : 0;
        // return sum(moves, playfieldIndex, FIELD_DIMENSION * FIELD_DIMENSION);
        
        __syncthreads();
        size_t sum = 0;
        for (int i = 0; i < _state->size; i++)
        {
            if (_state->possible[i])
            {
                sum++;
                __syncthreads();
            }
        }
        return sum;
    }

    __device__ size_t getRandomMoveIndex(size_t moveCount, float fakedRandom = -1)
    {
        size_t randomMoveIndex = 0;
        if (moveCount > 1)
        {
            if (fakedRandom >= 0)
            {
                randomMoveIndex = fakedRandom * moveCount;
            }
            else
            {
                randomMoveIndex = randomNumber(_deviceState, moveCount);    
            }
        }
        size_t possibleMoveIndex = 0;
        for (size_t i = 0; i < _state->size; ++i)
        {
            if (_state->possible[i])
            {
                if (possibleMoveIndex == randomMoveIndex)
                {
                    return i;
                }
                possibleMoveIndex++;;
            }
        }
        return 0;
    }

    __device__ void flipDirection(size_t moveIndex, int directionX, int directionY)
    {
        int currentIndex = _playfieldIndex;
        Player enemyPlayer = _state->getEnemyPlayer();
        bool flip = false;

        for (currentIndex = _playfieldIndex; _state->inBounds(currentIndex); currentIndex += directionY * _state->sideLength + directionX)
        {
            if(_state->field[currentIndex] != enemyPlayer)
            {
                flip = (_state->field[currentIndex] == _state->currentPlayer && currentIndex != _playfieldIndex);
                break;
            }
        }
        __syncthreads();
        if (flip)
        {
            for (; currentIndex - moveIndex != 0 ; currentIndex -= directionY * _state->sideLength + directionX)
            {
                _state->field[currentIndex] = _state->currentPlayer;
            }
        }
    }

    __device__ void flipEnemyCounter(size_t moveIndex)
    {
        int directionX = _playfieldX - moveIndex % _state->sideLength;
        int directionY = _playfieldY - moveIndex / _state->sideLength;

        if (abs(directionX) <= 1 && abs(directionY) <= 1 && moveIndex != _playfieldIndex)
        {
            flipDirection(moveIndex, directionX, directionY);
        }
    }
};



__device__ bool doStep(CudaGameState& state, CudaSimulator& simulator, float fakedRandom = -1)
{
    __syncthreads();

    simulator.calculatePossibleMoves();
    size_t moveCount = simulator.countPossibleMoves();

    if (moveCount > 0)
    {
        __shared__ size_t index;
        if (threadIdx.x == 0)
            index = simulator.getRandomMoveIndex(moveCount, fakedRandom);
        
        __syncthreads();

        simulator.flipEnemyCounter(index);

        __syncthreads();

        state.field[index] = state.currentPlayer;
    }
    state.currentPlayer = state.getEnemyPlayer();
    return moveCount > 0;
}

__global__ void simulateSingleStep(hiprandState* deviceState, Field* playfield, Player currentPlayer, float fakedRandom)
{
    int playfieldIndex = threadIdx.x;
    __shared__ Field sharedPlayfield[FIELD_DIMENSION * FIELD_DIMENSION];
    __shared__ bool possibleMoves[FIELD_DIMENSION*FIELD_DIMENSION];
    sharedPlayfield[playfieldIndex] = playfield[playfieldIndex];

    CudaGameState state =  { 
        sharedPlayfield, 
        possibleMoves, 
        FIELD_DIMENSION * FIELD_DIMENSION, 
        FIELD_DIMENSION, 
        currentPlayer 
    };
    CudaSimulator simulator(&state, deviceState);

    doStep(state, simulator, fakedRandom);

    playfield[playfieldIndex] = sharedPlayfield[playfieldIndex];
}


__global__ void simulateGameLeaf(hiprandState* deviceState, Field* playfield, Player currentPlayer)
{
    int playfieldIndex = threadIdx.x;

    __shared__ Field sharedPlayfield[FIELD_DIMENSION * FIELD_DIMENSION];
    __shared__ bool possibleMoves[FIELD_DIMENSION*FIELD_DIMENSION];
    sharedPlayfield[playfieldIndex] = playfield[playfieldIndex];

    CudaGameState state =  { 
        sharedPlayfield, 
        possibleMoves, 
        FIELD_DIMENSION * FIELD_DIMENSION, 
        FIELD_DIMENSION, 
        currentPlayer 
    };
    CudaSimulator simulator(&state, deviceState);
    size_t passCounter = 0;
    size_t limit = 64;
    while (limit > 0)
    {
        if (!doStep(state, simulator))
        {
            passCounter++;
            if (passCounter > 1)
                break;
        }
        else
        {
            passCounter = 0;
        }
        -- limit;
    }

    if (playfieldIndex == 0)
        printf("Runs: %d\n", limit);
    
    __syncthreads();
    
	playfield[playfieldIndex] = sharedPlayfield[playfieldIndex];
}