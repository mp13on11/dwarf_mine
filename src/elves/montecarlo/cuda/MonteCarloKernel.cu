#include "hip/hip_runtime.h"
#include "MonteCarloTreeSearch.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "OthelloField.h"
#include "CudaGameState.cuh"
#include "CudaMove.cuh"
#include "CudaSimulator.cuh"
#include "CudaUtil.cuh"
#include "CudaDebug.cuh"
#include <assert.h>

__global__ void setupStateForRandom(hiprandState* state, size_t* seeds)
{
	hiprand_init(seeds[blockIdx.x], 0, 0, &state[blockIdx.x]);
}


__global__ void setupStateForRandom(hiprandState* states, float* randomValues, size_t numberOfRandomValues)
{
    hiprand_init(threadIdx.x, 0, 0, &states[threadIdx.x]);
    for (size_t i = 0; i + threadIdx.x < numberOfRandomValues; i += 128)
    {
        hiprandState deviceState = states[threadIdx.x];
        randomValues[i + threadIdx.x] = 1.0f - hiprand_uniform(&deviceState); // delivers (0, 1] - we need [0, 1)
        states[threadIdx.x] = deviceState;
    }
}

__device__ bool doStep(CudaGameState& state, CudaSimulator& simulator, size_t limit, float fakedRandom = -1)
{
    cassert(state.size == FIELD_DIMENSION * FIELD_DIMENSION, "Block %d, Thread %d detected invalid field size of %li\n", blockIdx.x, threadIdx.x, state.size);
    
    simulator.calculatePossibleMoves();
    
    size_t moveCount = simulator.countPossibleMoves();
    
    if (moveCount > 0)
    {
        size_t index = simulator.getRandomMoveIndex(moveCount, fakedRandom);
        cassert(index < state.size, "Block %d, Thread %d: Round %d detected unexpected move index %d for maximal playfield size %lu\n", blockIdx.x, limit, index, state.size);

        simulator.flipEnemyCounter(index, limit);

        cassert(!state.isUnchanged(), "Block %d: %lu detected unchanged state\n", blockIdx.x, limit);
    }

    state.currentPlayer = state.getEnemyPlayer();
    return moveCount > 0;
}

__device__ void expandLeaf(CudaSimulator& simulator, CudaGameState& state)
{
    Player startingPlayer = state.currentPlayer;
    size_t passCounter = 0;
    size_t rounds = 0;

    __syncthreads();
    
    while (passCounter < 2)
    {
        bool passedMove = !doStep(state, simulator, rounds);
        passCounter = (passedMove ? passCounter + 1 : 0);

        cassert (rounds < MAXIMAL_MOVE_COUNT, "Detected rounds overflowing maximal count %d in %d\n", MAXIMAL_MOVE_COUNT, threadIdx.x); 
        rounds++;
    }
    __syncthreads();
}

__device__ void expandLeaf(hiprandState* deviceState, CudaSimulator& simulator, CudaGameState& state)
{
    size_t passCounter = 0;
    size_t rounds = 0;

    __syncthreads();
    
    while (passCounter < 2)
    {
        bool passedMove = !doStep(state, simulator, rounds);
        passCounter = (passedMove ? passCounter + 1 : 0);

        cassert (rounds < MAXIMAL_MOVE_COUNT, "Detected rounds overflowing maximal count %d in %d\n", MAXIMAL_MOVE_COUNT, threadIdx.x); 
        rounds++;
    }
    __syncthreads();
}

__global__ void simulateGame(size_t reiterations, hiprandState* deviceStates, size_t numberOfPlayfields, const Field* playfields, Player currentPlayer, OthelloResult* results)
{
    int playfieldIndex = threadIdx.x;

    for (size_t i = 0; i < reiterations; ++i)
    {
        size_t node = randomNumber(deviceStates, numberOfPlayfields);

        __shared__ Field sharedPlayfield[FIELD_DIMENSION * FIELD_DIMENSION];
        __shared__ Field oldPlayfield[FIELD_DIMENSION * FIELD_DIMENSION];
        __shared__ bool possibleMoves[FIELD_DIMENSION*FIELD_DIMENSION];
        
        size_t playfieldOffset = FIELD_DIMENSION * FIELD_DIMENSION * node;
        sharedPlayfield[playfieldIndex] = playfields[playfieldOffset + playfieldIndex];

        CudaGameState state =  { 
            sharedPlayfield, 
            oldPlayfield,
            possibleMoves, 
            FIELD_DIMENSION * FIELD_DIMENSION, 
            FIELD_DIMENSION, 
            currentPlayer 
        };
        CudaSimulator simulator(&state, deviceStates);

        __syncthreads();

        expandLeaf(deviceStates, simulator, state);
        
        __syncthreads();
        if (state.isWinner(currentPlayer))
        {
            if (threadIdx.x == 0)
                results[node].wins++;
        }
        if (threadIdx.x == 0)
        {
            results[node].visits ++;
        }
    }
}

__global__ void simulateGamePreRandom(size_t reiterations, size_t numberOfBlocks, float* randomValues, size_t numberOfPlayfields, const Field* playfields, Player currentPlayer, OthelloResult* results)
{
    int playfieldIndex = threadIdx.x;
    size_t blockIterations = size_t(ceil(reiterations * 1.0 / numberOfBlocks));

    for (size_t i = 0; i < blockIterations; ++i)
    {
		size_t randomSeed = i * numberOfBlocks + blockIdx.x;

        cassert(randomSeed < reiterations + 121, "SeedIndex %lu exceeded reiterations\n", randomSeed);
        size_t node = randomNumber(randomValues, &randomSeed, numberOfPlayfields);

        __shared__ Field sharedPlayfield[FIELD_DIMENSION * FIELD_DIMENSION];
        __shared__ Field oldPlayfield[FIELD_DIMENSION * FIELD_DIMENSION];
        __shared__ bool possibleMoves[FIELD_DIMENSION*FIELD_DIMENSION];
        
        size_t playfieldOffset = FIELD_DIMENSION * FIELD_DIMENSION * node;
        sharedPlayfield[playfieldIndex] = playfields[playfieldOffset + playfieldIndex];

        CudaGameState state =  { 
            sharedPlayfield, 
            oldPlayfield,
            possibleMoves, 
            FIELD_DIMENSION * FIELD_DIMENSION, 
            FIELD_DIMENSION, 
            currentPlayer
        };

        CudaSimulator simulator(&state, randomValues, randomSeed);

        __syncthreads();

        expandLeaf(simulator, state);
        
        __syncthreads();
        if (state.isWinner(currentPlayer))
        {
            if (threadIdx.x == 0)
                results[node].wins++;
        }
        if (threadIdx.x == 0)
        {
            results[node].visits ++;
        }
    }
}

__global__ void testRandomNumber(float fakedRandom, size_t maximum, size_t* randomNumberResult)
{
    *randomNumberResult = randomNumber(NULL, maximum, fakedRandom);
}

__global__ void testNumberOfMarkedFields(size_t* resultSum, const bool* playfield)
{
    *resultSum = numberOfMarkedFields(playfield);
}


__global__ void testDoStep(hiprandState* deviceState, Field* playfield, Player currentPlayer, float fakedRandom)
{
    int playfieldIndex = threadIdx.x;
    __shared__ Field sharedPlayfield[FIELD_DIMENSION * FIELD_DIMENSION];
    __shared__ Field oldPlayfield[FIELD_DIMENSION * FIELD_DIMENSION];
    __shared__ bool possibleMoves[FIELD_DIMENSION*FIELD_DIMENSION];
    sharedPlayfield[playfieldIndex] = playfield[playfieldIndex];

    // this part may be a shared variable?
    CudaGameState state =  { 
        sharedPlayfield, 
        oldPlayfield,
        possibleMoves, 
        FIELD_DIMENSION * FIELD_DIMENSION, 
        FIELD_DIMENSION, 
        currentPlayer 
    };
    CudaSimulator simulator(&state, deviceState);

    doStep(state, simulator, 0, fakedRandom);

    playfield[playfieldIndex] = sharedPlayfield[playfieldIndex];
}

__global__ void testExpandLeaf(hiprandState* deviceState, Field* playfield, Player currentPlayer, size_t* wins, size_t* visits)
{
    int playfieldIndex = threadIdx.x;

    __shared__ Field sharedPlayfield[FIELD_DIMENSION * FIELD_DIMENSION];
    __shared__ Field oldPlayfield[FIELD_DIMENSION * FIELD_DIMENSION];
    __shared__ bool possibleMoves[FIELD_DIMENSION*FIELD_DIMENSION];
    sharedPlayfield[playfieldIndex] = playfield[playfieldIndex];

    CudaGameState state =  { 
        sharedPlayfield, 
        oldPlayfield,
        possibleMoves, 
        FIELD_DIMENSION * FIELD_DIMENSION, 
        FIELD_DIMENSION, 
        currentPlayer 
    };
    CudaSimulator simulator(&state, deviceState);
    expandLeaf(deviceState, simulator, state);
    if (state.isWinner(currentPlayer))
    {
        if (threadIdx.x == 0) ++(*wins);
    }
    if (threadIdx.x == 0)
        (*visits)++;
	playfield[playfieldIndex] = sharedPlayfield[playfieldIndex];
}
